#include "hip/hip_runtime.h"
#include <librta/basic_types.h>
#include <librta/cuda-vec.h>
#include <librta/cuda-kernels.h>

#include <iostream>

using namespace rta;
using namespace std;

namespace rta {
	namespace cgls {
		
		namespace k {
			static __global__ void update_triangle_data(cuda::simple_triangle *tri, int N, int offset, 
												 float3 *v, float3 *n, float2 *t, uint3 *I, int triangles,
												 int wg_size, int work_groups, int iter, int3 *wg_data) {

				uint local_thread_id = threadIdx.x;
				int work_group_id = blockIdx.x;
				
				int3 wgd = wg_data[work_group_id];
				#define batch_offset   (wgd.x)
				#define batch_size     (wgd.y)
				#define batch_material (wgd.z)

				for (int i = 0; i < iter; ++i) {
					if (wg_size * i + local_thread_id >= batch_size)
						break;

					int tri_id = batch_offset    					// batch tri-offset depends on drawelement sizes, as not all batches are filled.
					           + wg_size * i						// iteration offset in batch
							   + local_thread_id;					// current triangle in sub-batch
					cuda::simple_triangle out;
					uint3 indices = I[tri_id];
					*((float3*)&out.a)  = v[indices.x];
					*((float3*)&out.b)  = v[indices.y];
					*((float3*)&out.c)  = v[indices.z];
					*((float3*)&out.na) = n[indices.x];
					*((float3*)&out.nb) = n[indices.y];
					*((float3*)&out.nc) = n[indices.z];
					*((float2*)&out.ta) = t[indices.x];
					*((float2*)&out.tb) = t[indices.y];
					*((float2*)&out.tc) = t[indices.z];
					out.material_index = batch_material;
					tri[tri_id] = out;
				}
				
			}
		}
		
		void update_triangle_data(basic_flat_triangle_list<cuda::simple_triangle> &ftl, int offset, 
								  float3 *v, float3 *n, float2 *t, uint3 *I, int triangles,
								  int wg_size, int iter, int work_groups, int3 *wg_data) {

			dim3 threads(wg_size);
			dim3 blocks = rta::cuda::block_configuration_2d(wg_size*work_groups, 1, threads);

			cout << "Conversion." << endl;
			cout << "  Triangles:   " << ftl.triangles << endl;
			cout << "  Offset:      " << offset << endl;
			cout << "  Triangles2:  " << triangles << endl;
			cout << "  wg_size:     " << wg_size << endl;
			cout << "  iter:        " << iter << endl;
			cout << "  work_groups: " << work_groups << endl;
			k::update_triangle_data<<<blocks,threads>>>(ftl.triangle, ftl.triangles, offset,
														v, n, t, I, triangles,
														wg_size, iter, work_groups/100, wg_data);
		}

	}
}
